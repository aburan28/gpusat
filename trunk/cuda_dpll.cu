#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include <hip/hip_runtime_api.h>
#include <sm_11_atomic_functions.h>
#include <sm_12_atomic_functions.h>

#define BUFF_SZ 255

#define NUM_BLOCK 1
#define NUM_THREADS 32

#define MAX_ASSIGN_SIZE 4096

char* partial_assign_cpu;
short* problem_cpu;				 // set of clauses
char* partial_assign_cuda;
short* problem_cuda;
int* flag_gpu ;
int* flag_cpu ;
unsigned int* result_gpu;
unsigned int* result_cpu;
//forward declarations
__host__ int numSat();
__host__ bool dpll_hlpr(unsigned int depth);
__host__ bool dpll();
__host__ bool cuda_dpll_hlpr(unsigned int depth);
__host__ bool cuda_dpll();

unsigned int nbclauses;
unsigned int nbvar;

__global__ void cuda_sat_eval (char* partial_assign, unsigned int num_clauses ,short* problem, unsigned int assign_size, int* flag_gpu , unsigned int* result_gpu)
{

	unsigned char v1,v2,v3;
	__shared__ unsigned short passign[MAX_ASSIGN_SIZE];
	int myid=blockIdx.x*NUM_THREADS+threadIdx.x;

	//copy from global partial assign to shared

	int blocksize=assign_size/NUM_THREADS+1;

	for(int i=1;i<=blocksize;i++)
	{
		if((myid*blocksize+i) < assign_size)
		{
			passign[myid*blocksize+i]=partial_assign[myid*blocksize+i];
//			printf(" partial assign index %d , cpu data %d , gpu data %d \n", myid*blocksize+i, partial_assign[myid*blocksize+i], passign[myid*blocksize+i]);
		}
	}
	__syncthreads();
	//iterate over problem blocked by thread.
	//each thread handles a few clauses
	int clauses_per_thread = num_clauses/(NUM_BLOCK*NUM_THREADS);

	__shared__ unsigned int clause_count[NUM_THREADS];

	clause_count[threadIdx.x] = 0 ;
	
	for(int ii=myid;ii<num_clauses;ii+=NUM_THREADS)
	{
		v1=(passign[abs(problem[3*ii])]^((problem[3*ii]>>(15))&1));
		v2=(passign[abs(problem[3*ii+1])]^((problem[3*ii+1]>>(15))&1));
		v3=(passign[abs(problem[3*ii+2])]^((problem[3*ii+2]>>(15))&1));

		if((v1|v2|v3)==0)
		{
			clause_count[threadIdx.x]= 0;
//			printf("v1 %d v2 %d v3 %d\n",v1,v2,v3);
//			printf("a1 %d a2 %d a3 %d\n",passign[abs(problem[3*ii])], passign[abs(problem[3*ii+1])], passign[abs(problem[3*ii+2])]);
//			printf("p1 %d p2 %d p3 %d\n",problem[3*ii],problem[3*ii+1],problem[3*ii+2]);
			*flag_gpu = -1;		 //unsatisfiable with this partial assignment
		}
		else if(v1==1||v2==1||v3==1)
		{
			clause_count[threadIdx.x] += 1;

		}
	}

	__syncthreads();

/*	if(threadIdx.x == 0)
	{
		printf("\n Clause Count Values \n");
		for (int tempint = 0 ; tempint < NUM_THREADS ; tempint ++)
		{
			printf ("index  %d , Value %d \n", tempint , clause_count[tempint]);
		}
	}
*/
//	printf ("Reduction\n");
	for (int k=(NUM_THREADS/2); k>0; k=k>>1)
	{
		if (threadIdx.x < k)
		{
			clause_count[threadIdx.x] += clause_count[ threadIdx.x + k ];
		//	printf("%d = %d + %d \n" , threadIdx.x , threadIdx.x , threadIdx.x + k);
		}
	__syncthreads();
	}
	__syncthreads();

	if(threadIdx.x == 0)
	{
//		printf(" Hi I am block %d , and my count is %d \n ", blockIdx.x , clause_count[0]);
		atomicAdd(result_gpu,clause_count[0]);
	}

}


__host__ void compareFail()
{
	//debug prinfs here
	printf("numsat for cuda and cpu differed\n");
	exit(1);
}


__host__ bool dpll_hlpr(unsigned int depth)
{
	int mynumsat;

	partial_assign_cpu[depth]=1;

	mynumsat=numSat();

	//printf("depth %d val %d numsat %d\n", depth, partial_assign_cpu[depth], mynumsat);

	if(mynumsat==nbvar)
		return true;

	if(mynumsat!=-1)
	{
		if(dpll_hlpr(depth+1))
			return true;
	}

	partial_assign_cpu[depth]=0;

	mynumsat=numSat();

	//printf("depth %d val %d numsat %d\n", depth, partial_assign_cpu[depth], mynumsat);

	if(mynumsat==nbvar)
		return true;

	if(mynumsat!=-1)
	{
		if(dpll_hlpr(depth+1))
			return true;
	}

	partial_assign_cpu[depth]=2;
	return false;

}


__host__ bool test_cuda_dpll_hlpr(unsigned int depth)
{
	int mynumsat,mynumsat2;

	partial_assign_cpu[depth]=1;

	mynumsat=numSat();
	// Cuda Kernel Call
	hipMemcpy(partial_assign_cuda+sizeof(char), partial_assign_cpu+sizeof(char), nbvar*sizeof(char), hipMemcpyHostToDevice);
	hipMemset(result_gpu, 0 ,sizeof(int));
	hipMemset(flag_gpu, 0 ,sizeof(int));

	cuda_sat_eval<<<NUM_BLOCK,NUM_THREADS>>>(partial_assign_cuda, nbclauses , problem_cuda, nbvar+1, flag_gpu, result_gpu);

	hipMemcpy(result_cpu,result_gpu, sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(flag_cpu, flag_gpu, sizeof(int),hipMemcpyDeviceToHost);

	if(*flag_cpu == -1)
		mynumsat2 = -1;
	else
		mynumsat2 = *result_cpu;

	printf("depth %d val %d numsat %d\n", depth, partial_assign_cpu[depth], mynumsat);
	if(mynumsat!=mynumsat2)
	{
		printf("cpu numsat %d, gpu numsat %d\n",mynumsat,mynumsat2);
		compareFail();
	}

	if(mynumsat==nbvar)
		return true;

	if(mynumsat!=-1)
	{
		if(test_cuda_dpll_hlpr(depth+1))
			return true;
	}

	partial_assign_cpu[depth]=0;

	mynumsat=numSat();
	// Cuda Kernel Call
	hipMemcpy(partial_assign_cuda+sizeof(char), partial_assign_cpu+sizeof(char), nbvar*sizeof(char), hipMemcpyHostToDevice);
	hipMemset(result_gpu, 0 ,sizeof(int));
	hipMemset(flag_gpu, 0 ,sizeof(int));
	cuda_sat_eval<<<NUM_BLOCK,NUM_THREADS>>>(partial_assign_cuda, nbclauses , problem_cuda, nbvar+1, flag_gpu, result_gpu  );

	hipMemcpy(result_cpu,result_gpu, sizeof(unsigned int),hipMemcpyDeviceToHost);
	hipMemcpy(flag_cpu, flag_gpu, sizeof(int),hipMemcpyDeviceToHost);

	if(*flag_cpu == -1)
		mynumsat2 = -1;
	else
		mynumsat2 = *result_cpu;

	printf("depth %d val %d numsat %d\n", depth, partial_assign_cpu[depth], mynumsat);
	if(mynumsat!=mynumsat2)
	{
		printf("cpu numsat %d, gpu numsat %d\n",mynumsat,mynumsat2);
		compareFail();
	}

	if(mynumsat==nbvar)
		return true;

	if(mynumsat!=-1)
	{
		if(test_cuda_dpll_hlpr(depth+1))
			return true;
	}

	partial_assign_cpu[depth]=2;

	return false;

}


__host__ bool cuda_dpll_hlpr(unsigned int depth)
{
	int mynumsat;

	partial_assign_cpu[depth]=1;

	// Cuda Kernel Call
	hipMemcpy(partial_assign_cuda+sizeof(char), partial_assign_cpu+sizeof(char), nbvar*sizeof(char), hipMemcpyHostToDevice);
	hipMemset(result_gpu, 0 ,sizeof(int));
	hipMemset(flag_gpu, 0 ,sizeof(int));

	cuda_sat_eval<<<NUM_BLOCK,NUM_THREADS>>>(partial_assign_cuda, nbclauses , problem_cuda, nbvar+1, flag_gpu, result_gpu);

	hipMemcpy(result_cpu,result_gpu, sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(flag_cpu, flag_gpu, sizeof(int),hipMemcpyDeviceToHost);

	if(*flag_cpu == -1)
		mynumsat = -1;
	else
		mynumsat = *result_cpu;


	if(mynumsat==nbvar)
		return true;

	if(mynumsat!=-1)
	{
		if(cuda_dpll_hlpr(depth+1))
			return true;
	}

	partial_assign_cpu[depth]=0;

	// Cuda Kernel Call
	hipMemcpy(partial_assign_cuda+sizeof(char), partial_assign_cpu+sizeof(char), nbvar*sizeof(char), hipMemcpyHostToDevice);
	hipMemset(result_gpu, 0 ,sizeof(int));
	hipMemset(flag_gpu, 0 ,sizeof(int));
	cuda_sat_eval<<<NUM_BLOCK,NUM_THREADS>>>(partial_assign_cuda, nbclauses , problem_cuda, nbvar+1, flag_gpu, result_gpu  );

	hipMemcpy(result_cpu,result_gpu, sizeof(unsigned int),hipMemcpyDeviceToHost);
	hipMemcpy(flag_cpu, flag_gpu, sizeof(int),hipMemcpyDeviceToHost);

	if(*flag_cpu == -1)
		mynumsat = -1;
	else
		mynumsat = *result_cpu;

	if(mynumsat==nbvar)
		return true;

	if(mynumsat!=-1)
	{
		if(cuda_dpll_hlpr(depth+1))
			return true;
	}

	partial_assign_cpu[depth]=2;

	return false;

}


__host__ bool cuda_dpll()
{
	bool rval;

	partial_assign_cpu = (char*)malloc((nbvar+1)*sizeof(char));
	memset(partial_assign_cpu, 2, nbvar);

	hipMalloc((void **) &partial_assign_cuda ,(nbvar+1)*sizeof(char));
	//hipMemset(partial_assign_cuda, 2 ,nbvar); no need to memset, we copy every time

	printf("functionality test of cuda dpll -- NOT FOR TIMING\n");
	rval = test_cuda_dpll_hlpr(1);
	hipFree(partial_assign_cuda);
	free(partial_assign_cpu);

	return rval;
}


__host__ bool dpll()
{
	bool rval;

	partial_assign_cpu = (char*)malloc((nbvar+1)*sizeof(char));
	memset(partial_assign_cpu, 2, nbvar);

	rval = dpll_hlpr(1);
	hipFree(partial_assign_cuda);
	free(partial_assign_cpu);

	return rval;
}


__host__ int numSat()
{
	unsigned int ii;
	unsigned int clause_count=0;
	unsigned char v1,v2,v3;

	for(ii=0;ii<nbclauses;ii++)
	{
		v1=(partial_assign_cpu[abs(problem_cpu[3*ii])]^((problem_cpu[3*ii]>>(15))&1));
		v2=(partial_assign_cpu[abs(problem_cpu[3*ii+1])]^((problem_cpu[3*ii+1]>>(15))&1));
		v3=(partial_assign_cpu[abs(problem_cpu[3*ii+2])]^((problem_cpu[3*ii+2]>>(15))&1));

		if((v1|v2|v3)==0)
			return -1;			 //unsatisfiable with this partial assignment
		else if(v1==1||v2==1||v3==1)
			clause_count++;
	}
	return clause_count;
}


int main(int argc, char** argv)
{

	char buff[BUFF_SZ];
	char linestart;
	unsigned int clause, ii;
	bool error;
	int read;

	memset(buff, '\0', BUFF_SZ);

	printf("c Parsing input\n");

	error=(EOF==scanf("%c%s%u%u", &linestart, buff, &nbvar, &nbclauses));
	error|=(linestart!='p');
	error|=(0!=strcmp(buff, "cnf"));

	if(error)
	{
		fprintf(stderr, "Cannot parse input file\n");
		exit(1);
	}

	problem_cpu = (short *)malloc(3*nbclauses*sizeof(short));
	result_cpu = (unsigned int *)malloc(sizeof(unsigned int));
	flag_cpu = (int *)malloc(sizeof(int));
	if(NULL==problem_cpu)
	{
		fprintf(stderr, "cannot malloc problem_cpu\n");
		exit(2);
	}
	ii=0;
	for(clause=0; clause<nbclauses; clause++)
	{
		do
		{
			if(EOF==scanf("%d",&read))
			{
				fprintf(stderr, "Cannot parse input file\n");
				exit(1);
			}
			if(read!=0)
			{
				problem_cpu[ii]=read;
				ii++;
			}

		}while(read!=0);

	}

	printf("c %u vars %u clauses\n", nbvar, nbclauses);
	// Copying the problem into CUDA memory

	hipMalloc((void **) &problem_cuda ,(3*nbclauses*sizeof(short)+3));
	hipMalloc((void **) &flag_gpu, (sizeof(int)));
	hipMalloc((void **) &result_gpu ,(sizeof(unsigned int)));
	hipMemcpy(problem_cuda, problem_cpu, 3*nbclauses*sizeof(short), hipMemcpyHostToDevice);

	if(cuda_dpll())
		printf("cpu: SATISFIABLE\n");
	else
	{
		printf("CPU: UNSATISFIABLE\n");
		//	printf("c tested %u of %u possible assignments\n",numtested,1<<nbvar);
	}

	//  Free Cuda memory
	hipFree(result_gpu);
	hipFree(problem_cuda);
	return 0;
}
